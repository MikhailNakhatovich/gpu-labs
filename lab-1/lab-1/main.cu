#include "hip/hip_runtime.h"
#include <algorithm>
#include <cinttypes>
#include <cstddef>
#include <iostream>
#include <new>
#include <random>
#include <windows.h>

#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 32
#define MINIMUM_VALUE -32
#define MAXIMUM_VALUE 32
#define DEFAULT_SIZE 1000

double* generateMatrix(size_t size)
{
    std::random_device device;
    std::mt19937 generator(device());
    std::uniform_real_distribution<double> distribution(MINIMUM_VALUE, MAXIMUM_VALUE);
    
    size_t n = size * size;
    double* matrix = new double[n];
    for (size_t i = 0; i < n; matrix[i] = distribution(generator), ++i);
    return matrix;
}

float multMatricesOnCPU(double* a, double* b, double* c, size_t size)
{
    LARGE_INTEGER startTime, stopTime, freq;
    QueryPerformanceFrequency(&freq);

    QueryPerformanceCounter(&startTime);
    for (size_t i = 0; i < size; ++i)
    {
        for (size_t j = 0; j < size; ++j)
        {
            size_t cellIndex = i * size + j;
            c[cellIndex] = 0;
            for (size_t k = 0; k < size; c[cellIndex] += a[i * size + k] * b[k * size + j], ++k);
        }
    }
    QueryPerformanceCounter(&stopTime);

    size_t timeDelta = stopTime.QuadPart - startTime.QuadPart;
    return static_cast<float>(timeDelta) / freq.QuadPart;
}

__global__ void multMatricesOnGPUKernel(double* a, double* b, double* c, size_t size)
{
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= size || j >= size)
        return;

    size_t cellIndex = i * size + j;
    c[cellIndex] = 0;
    for (size_t k = 0; k < size; c[cellIndex] += a[i * size + k] * b[k * size + j], ++k);
}

float multMatricesOnGPU(double* a, double* b, double* c, size_t size)
{
    double* adev, *bdev, *cdev;
    size_t numBytes = size * size * sizeof(double);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks((size + threads.x - 1) / threads.x, (size + threads.y - 1) / threads.y);

    hipMalloc(reinterpret_cast<void**>(&adev), numBytes);
    hipMalloc(reinterpret_cast<void**>(&bdev), numBytes);
    hipMalloc(reinterpret_cast<void**>(&cdev), numBytes);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

    multMatricesOnGPUKernel<<<blocks, threads>>>(adev, bdev, cdev, size);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);

    return gpuTime / 1000.0f;
}

double getMaximumDeviation(double* a, double* b, size_t size)
{
    size_t n = size * size;
    double deviation = 0.0;
    for (size_t i = 0; i < n; deviation = std::max(deviation, std::abs(a[i] - b[i])), ++i);
    return deviation;
}

int main(int argc, char* argv[])
{
    size_t size = DEFAULT_SIZE;
    if (argc > 1)
        size = std::strtoumax(argv[1], nullptr, 10);
    std::cout << "Matrices have size " << size << "x" << size << std::endl;

    double* a = generateMatrix(size), *b = generateMatrix(size);
    double* cCPU = new double[size * size], *cGPU = new double[size * size];

    float timeCPU = multMatricesOnCPU(a, b, cCPU, size);
    float timeGPU = multMatricesOnGPU(a, b, cGPU, size);

    std::cout << "Elapsed times:" << std::endl;
    std::cout << "CPU: " << timeCPU << " seconds" << std::endl;
    std::cout << "GPU: " << timeGPU << " seconds" << std::endl;
    std::cout << "Maximum deviation between result matrices equals to " << getMaximumDeviation(cCPU, cGPU, size) << std::endl;

    delete[] a;
    delete[] b;
    delete[] cCPU;
    delete[] cGPU;

    return 0;
}
